#include <iostream>
#include <hip/hip_runtime.h>
__global__ void VectorAddition(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 10;
    float A[N], B[N], C[N];

    for(int i = 0; i< N; i++){
        A[i] = i;
        B[i] = i * 2;
    }

    float *d_a, *d_b,*d_c;
    hipMalloc(&d_a,N*sizeof(float));
    hipMalloc(&d_b,N*sizeof(float));
    hipMalloc(&d_c,N*sizeof(float));
    hipMemcpy(d_a,A,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_b,B,N*sizeof(float),hipMemcpyHostToDevice);
    int blocksize=256;
    int gridsize=ceil(N/blocksize);
    VectorAddition<<<gridsize,blocksize>>>(d_a,d_b,d_c,N);
     
    hipDeviceSynchronize();

    hipMemcpy(C,d_c,N*sizeof(float),hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++){
        std::cout << A[i] << " + " << B[i] << " = " << C[i] << std::endl;
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
     
    return 0;
}