#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

#define N 4

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if(err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << "@" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

__global__ void matVecMul(const float *A, const float *x, float *y, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n) {
        float sum = 0.0f;
        for (int j = 0; j < n; j++) {
            sum += A[row * n + j] * x[j];
        }
        y[row] = sum;
    }
}

__global__ void vecAdd(float *y, const float *x, float alpha, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] += alpha * x[i];
    }
}

__global__ void vecSub(float *y, const float *x, float alpha, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] -= alpha * x[i];
    }
}

__global__ void vecScale(float *x, float beta, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        x[i] *= beta;
    }
}

__global__ void dotProduct(const float *a, const float *b, float *result, int n) {
    __shared__ float cache[256];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float temp = 0.0f;
    while (i < n) {
        temp += a[i] * b[i];
        i += blockDim.x * gridDim.x;
    }
    cache[tid] = temp;
    __syncthreads();

    int blockSize = blockDim.x;
    while (blockSize > 1) {
        int halfPoint = blockSize / 2;
        if (tid < halfPoint) {
            cache[tid] += cache[tid + halfPoint];
        }
        __syncthreads();
        blockSize = halfPoint;
    }
    if (tid == 0) {
        atomicAdd(result, cache[0]);
    }
}

int main() {
    const int n = N;
    const int matrixSize = n * n * sizeof(float);
    const int vectorSize = n * sizeof(float);

    float h_A[N * N] = {
         4, 1, 0, 0,
         1, 3, 1, 0,
         0, 1, 2, 1,
         0, 0, 1, 1
    };
    float h_b[N] = {15, 10, 10, 10};
    float h_x[N] = {0}; 

    float *d_A, *d_x, *d_b, *d_r, *d_p, *d_Ap;
    CUDA_CHECK(hipMalloc(&d_A, matrixSize));
    CUDA_CHECK(hipMalloc(&d_x, vectorSize));
    CUDA_CHECK(hipMalloc(&d_b, vectorSize));
    CUDA_CHECK(hipMalloc(&d_r, vectorSize));
    CUDA_CHECK(hipMalloc(&d_p, vectorSize));
    CUDA_CHECK(hipMalloc(&d_Ap, vectorSize));

    CUDA_CHECK(hipMemcpy(d_A, h_A, matrixSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b, vectorSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_x, h_x, vectorSize, hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpy(d_r, d_b, vectorSize, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(d_p, d_r, vectorSize, hipMemcpyDeviceToDevice));

    float *d_dot;
    CUDA_CHECK(hipMalloc(&d_dot, sizeof(float)));

    float rdotr = 0.0f, new_rdotr = 0.0f;

    CUDA_CHECK(hipMemset(d_dot, 0, sizeof(float)));
    dotProduct<<<1, 256>>>(d_r, d_r, d_dot, n);
    CUDA_CHECK(hipMemcpy(&rdotr, d_dot, sizeof(float), hipMemcpyDeviceToHost));

    int max_iter = 1000;
    float tol = 1e-6f;
    int k = 0;

    while (sqrt(rdotr) > tol && k < max_iter) {
   
        matVecMul<<<(n + 255) / 256, 256>>>(d_A, d_p, d_Ap, n);

        CUDA_CHECK(hipMemset(d_dot, 0, sizeof(float)));
        dotProduct<<<1, 256>>>(d_p, d_Ap, d_dot, n);
        float pAp = 0.0f;
        CUDA_CHECK(hipMemcpy(&pAp, d_dot, sizeof(float), hipMemcpyDeviceToHost));

        float alpha = rdotr / pAp;

        vecAdd<<<(n + 255) / 256, 256>>>(d_x, d_p, alpha, n);

        vecSub<<<(n + 255) / 256, 256>>>(d_r, d_Ap, alpha, n);

        CUDA_CHECK(hipMemset(d_dot, 0, sizeof(float)));
        dotProduct<<<1, 256>>>(d_r, d_r, d_dot, n);
        CUDA_CHECK(hipMemcpy(&new_rdotr, d_dot, sizeof(float), hipMemcpyDeviceToHost));

        if (sqrt(new_rdotr) < tol) {
            break;
        }

        float beta = new_rdotr / rdotr;

        vecScale<<<(n + 255) / 256, 256>>>(d_p, beta, n);
        
        vecAdd<<<(n + 255) / 256, 256>>>(d_p, d_r, 1.0f, n);

        rdotr = new_rdotr;
        k++;
    }

    CUDA_CHECK(hipMemcpy(h_x, d_x, vectorSize, hipMemcpyDeviceToHost));
    std::cout << "Conjugate Gradient converged in " << k << " iterations." << std::endl;
    std::cout << "Solution x:" << std::endl;
    for (int i = 0; i < n; i++) {
        std::cout << h_x[i] << std::endl;
    }

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_r));
    CUDA_CHECK(hipFree(d_p));
    CUDA_CHECK(hipFree(d_Ap));
    CUDA_CHECK(hipFree(d_dot));

    return 0;
}