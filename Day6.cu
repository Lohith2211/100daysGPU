#include <hip/hip_runtime.h>
#include <iostream>

// Dimensions of the matrix
constexpr int MATRIX_WIDTH = 1024;
constexpr int MATRIX_HEIGHT = 1024;

// CUDA kernel to transpose a matrix
__global__ void transposeKernel(const float* src, float* dst, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        int srcIdx = row * width + col;
        int dstIdx = col * height + row;
        dst[dstIdx] = src[srcIdx];
    }
}

// Utility function for checking CUDA API errors
void verifyCudaCall(const char* context) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << context << ": CUDA error: " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

int main() {
    int width = MATRIX_WIDTH;
    int height = MATRIX_HEIGHT;
    size_t totalBytes = width * height * sizeof(float);

    // Allocate and initialize host memory
    float* hostSrc = (float*)malloc(totalBytes);
    float* hostDst = (float*)malloc(totalBytes);
    for (int i = 0; i < width * height; ++i) {
        hostSrc[i] = static_cast<float>(i);
    }

    // Allocate device buffers
    float *deviceSrc, *deviceDst;
    hipMalloc(&deviceSrc, totalBytes);
    hipMalloc(&deviceDst, totalBytes);

    // Copy input matrix from host to device
    hipMemcpy(deviceSrc, hostSrc, totalBytes, hipMemcpyHostToDevice);
    verifyCudaCall("Memcpy host to device");

    // Define CUDA thread block and grid dimensions
    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid(
        (width + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (height + threadsPerBlock.y - 1) / threadsPerBlock.y
    );

    // Launch kernel to transpose matrix
    transposeKernel<<<blocksPerGrid, threadsPerBlock>>>(deviceSrc, deviceDst, width, height);
    hipDeviceSynchronize();
    verifyCudaCall("Kernel execution");

    // Copy result back from device to host
    hipMemcpy(hostDst, deviceDst, totalBytes, hipMemcpyDeviceToHost);
    verifyCudaCall("Memcpy device to host");

    // Validate the transposed matrix
    bool isCorrect = true;
    for (int col = 0; col < width && isCorrect; ++col) {
        for (int row = 0; row < height; ++row) {
            if (hostDst[col * height + row] != hostSrc[row * width + col]) {
                isCorrect = false;
                break;
            }
        }
    }

    std::cout << (isCorrect ? "Transpose successful!" : "Transpose verification failed!") << std::endl;

    // Cleanup resources
    hipFree(deviceSrc);
    hipFree(deviceDst);
    free(hostSrc);
    free(hostDst);

    return 0;
}
