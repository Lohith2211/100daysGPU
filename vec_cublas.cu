
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

int main() {
    const int N = 10;
    float A[N], B[N], C[N];

    for(int i = 0; i < N; i++) {
        A[i] = i;
        B[i] = i;
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float *d_a, *d_b;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));

    hipMemcpy(d_a, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, N * sizeof(float), hipMemcpyHostToDevice);

    const float alpha = 1.0f;

    hipblasSaxpy(handle, N, &alpha, d_a, 1, d_b, 1);

    hipMemcpy(C, d_b, N * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++) {
        std::cout << C[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipblasDestroy(handle);

    return 0;
}