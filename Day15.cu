#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <limits>

#include "helper.cuh"
#include "kernels.cuh"

#define BLOCK_SIZE 1024
#define THREADS_PER_BLOCK 1024
#define NEGATIVE_INFINITY -1e38f

void flashAttention2BackwardPass(
    const float* query, const float* key, const float* value,
    const float* output, const float* dOutput,
    float* dQuery, float* dKey, float* dValue,
    int numTokens, int dim, int blockCols, int blockRows,
    float* hostL
) {
    float scale = 1.0f / sqrtf((float)dim);

    float* deviceD;
    hipMalloc((void**)&deviceD, numTokens * sizeof(float));
    computeDKernel<<<(numTokens + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
        dOutput, output, deviceD, numTokens, dim);
    hipDeviceSynchronize();

    float* hostD = (float*)malloc(numTokens * sizeof(float));
    hipMemcpy(hostD, deviceD, numTokens * sizeof(float), hipMemcpyDeviceToHost);

    hipMemset(dQuery, 0, numTokens * dim * sizeof(float));
    hipMemset(dKey, 0, numTokens * dim * sizeof(float));
    hipMemset(dValue, 0, numTokens * dim * sizeof(float));

    for (int colBlockIdx = 0; colBlockIdx < (numTokens + blockCols - 1) / blockCols; ++colBlockIdx) {
        float* hostKeyBlock = (float*)malloc(blockCols * dim * sizeof(float));
        float* hostValueBlock = (float*)malloc(blockCols * dim * sizeof(float));

        hipMemcpy(hostKeyBlock, key + colBlockIdx * blockCols * dim, blockCols * dim * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(hostValueBlock, value + colBlockIdx * blockCols * dim, blockCols * dim * sizeof(float), hipMemcpyDeviceToHost);

        float *deviceKeyBlock, *deviceValueBlock;
        hipMalloc((void**)&deviceKeyBlock, blockCols * dim * sizeof(float));
        hipMalloc((void**)&deviceValueBlock, blockCols * dim * sizeof(float));
        hipMemcpy(deviceKeyBlock, hostKeyBlock, blockCols * dim * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(deviceValueBlock, hostValueBlock, blockCols * dim * sizeof(float), hipMemcpyHostToDevice);

        float *tempGradKeyBlock, *tempGradValueBlock;
        hipMalloc((void**)&tempGradKeyBlock, blockCols * dim * sizeof(float));
        hipMalloc((void**)&tempGradValueBlock, blockCols * dim * sizeof(float));
        hipMemset(tempGradKeyBlock, 0, blockCols * dim * sizeof(float));
        hipMemset(tempGradValueBlock, 0, blockCols * dim * sizeof(float));

        for (int rowBlockIdx = 0; rowBlockIdx < (numTokens + blockRows - 1) / blockRows; ++rowBlockIdx) {
            const float* queryBlock = query + rowBlockIdx * blockRows * dim;
            const float* dOutputBlock = dOutput + rowBlockIdx * blockRows * dim;
            float* tempGradQueryBlock;
            hipMalloc((void**)&tempGradQueryBlock, blockRows * dim * sizeof(float));
            hipMemset(tempGradQueryBlock, 0, blockRows * dim * sizeof(float));

            const float* LBlock = hostL + rowBlockIdx * blockRows;
            const float* DBlock = hostD + rowBlockIdx * blockRows;

            float *deviceSBlock, *devicePBlock, *deviceDPBlock, *deviceDSBlock;
            hipMalloc((void**)&deviceSBlock, blockRows * blockCols * sizeof(float));
            hipMalloc((void**)&devicePBlock, blockRows * blockCols * sizeof(float));
            hipMalloc((void**)&deviceDPBlock, blockRows * blockCols * sizeof(float));
            hipMalloc((void**)&deviceDSBlock, blockRows * blockCols * sizeof(float));

            float* deviceMaxSBlock;
            hipMalloc((void**)&deviceMaxSBlock, blockRows * sizeof(float));

            computeSiKernel<<<(blockRows + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
                queryBlock, deviceKeyBlock, deviceSBlock, blockRows, blockCols, dim, scale);
            hipDeviceSynchronize();

            findRowMaxSiKernel<<<(blockRows + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
                deviceSBlock, deviceMaxSBlock, blockRows, blockCols);
            hipDeviceSynchronize();

            computePiKernel<<<(blockRows + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
                deviceSBlock, LBlock, devicePBlock, blockRows, blockCols, deviceMaxSBlock);
            hipDeviceSynchronize();

            computeDViKernel<<<(dim + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
                devicePBlock, dOutputBlock, tempGradValueBlock, blockRows, blockCols, dim);
            hipDeviceSynchronize();

            computeDPiKernel<<<(blockRows + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
                dOutputBlock, deviceValueBlock, deviceDPBlock, blockRows, blockCols, dim);
            hipDeviceSynchronize();

            computeDSiKernel<<<(blockRows + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
                devicePBlock, deviceDPBlock, DBlock, deviceDSBlock, blockRows, blockCols);
            hipDeviceSynchronize();

            computeDQiKernel<<<(blockRows + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
                deviceDSBlock, deviceKeyBlock, tempGradQueryBlock, blockRows, dim, blockCols);
            hipDeviceSynchronize();

            computeDKjKernel<<<(blockCols + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
                deviceDSBlock, queryBlock, tempGradKeyBlock, blockCols, dim, blockRows);
            hipDeviceSynchronize();

            accumulateDQKernel<<<(blockRows * dim + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
                dQuery, tempGradQueryBlock, blockRows, dim, rowBlockIdx * blockRows * dim);
            hipDeviceSynchronize();

            hipFree(deviceSBlock);
            hipFree(devicePBlock);
            hipFree(deviceDPBlock);
            hipFree(deviceDSBlock);
            hipFree(deviceMaxSBlock);
            hipFree(tempGradQueryBlock);
        }

        accumulateDKVjKernel<<<(blockCols * dim + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
            dKey, dValue, tempGradKeyBlock, tempGradValueBlock, blockCols, dim, colBlockIdx * blockCols * dim);
        hipDeviceSynchronize();

        hipFree(deviceKeyBlock);
        hipFree(deviceValueBlock);
        hipFree(tempGradKeyBlock);
        hipFree(tempGradValueBlock);
        free(hostKeyBlock);
        free(hostValueBlock);
    }

    hipFree(deviceD);
    free(hostD);
}
