#include <hip/hip_runtime.h>
#include <math.h>

__device__ void apply_rotary_embedding(
    float* q,           
    float* k,           
    const int head_dim, 
    const int position, 
    const float base = 10000.0f
) {

    for (int i = 0; i < head_dim; i += 2) {
        float freq = 1.0f / powf(base, (float)(i) / head_dim);
        float theta = position * freq;
        
        float cos_theta = cosf(theta);
        float sin_theta = sinf(theta);
        
        float q_real = q[i];
        float q_img = q[i + 1];
        float k_real = k[i];
        float k_img = k[i + 1];
        
        q[i] = q_real * cos_theta - q_img * sin_theta;
        q[i + 1] = q_real * sin_theta + q_img * cos_theta;
        
        k[i] = k_real * cos_theta - k_img * sin_theta;
        k[i + 1] = k_real * sin_theta + k_img * cos_theta;
    }
}

__global__ void rope_kernel(
    float* queries,       
    float* keys,         
    const int batch_size,
    const int seq_len,
    const int num_heads,
    const int head_dim
) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int batch_idx = idx / (seq_len * num_heads);
    int seq_idx = (idx / num_heads) % seq_len;
    int head_idx = idx % num_heads;
    
    if (batch_idx >= batch_size) return;
    
    int base_idx = batch_idx * (seq_len * num_heads * head_dim) + 
                   seq_idx * (num_heads * head_dim) +
                   head_idx * head_dim;
    
    apply_rotary_embedding(
        &queries[base_idx],
        &keys[base_idx],
        head_dim,
        seq_idx
    );
}

void apply_rope(
    float* d_queries,
    float* d_keys,
    const int batch_size,
    const int seq_len,
    const int num_heads,
    const int head_dim
) {
    dim3 block_size(256);
    dim3 grid_size((batch_size * seq_len * num_heads + block_size.x - 1) / block_size.x);
    
    rope_kernel<<<grid_size, block_size>>>(
        d_queries,
        d_keys,
        batch_size,
        seq_len,
        num_heads,
        head_dim
    );
} 