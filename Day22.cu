#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

constexpr int NUM_CLUSTERS = 2;
constexpr int N = 1024;
constexpr int THREADS_PER_BLOCK = 256;

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA Error: " << hipGetErrorString(error) \
                      << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

__global__ void eStepKernel(float* data, int N, float* mu, float* sigma, 
                            float* pival, float* responsibilities) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float x = data[idx];
        float probs[NUM_CLUSTERS];
        float sum = 0.0f;

        for (int k = 0; k < NUM_CLUSTERS; k++) {
            float diff = x - mu[k];
            float exponent = -0.5f * (diff * diff) / (sigma[k] * sigma[k]);
            float gauss = (1.0f / (sqrtf(2.0f * M_PI) * sigma[k])) * expf(exponent);
            probs[k] = pival[k] * gauss;
            sum += probs[k];
        }

        for (int k = 0; k < NUM_CLUSTERS; k++) {
            responsibilities[idx * NUM_CLUSTERS + k] = probs[k] / sum;
        }
    }
}

__global__ void mStepKernel(float* data, int N, float* responsibilities,
                            float* sum_gamma, float* sum_x, float* sum_x2) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float x = data[idx];
        for (int k = 0; k < NUM_CLUSTERS; k++) {
            float gamma = responsibilities[idx * NUM_CLUSTERS + k];
            atomicAdd(&sum_gamma[k], gamma);
            atomicAdd(&sum_x[k], gamma * x);
            atomicAdd(&sum_x2[k], gamma * x * x);
        }
    }
}

int main() {
    srand(static_cast<unsigned>(time(NULL)));

    float h_data[N];
    for (int i = 0; i < N; i++) {
        if (i < N / 2) {
            h_data[i] = 2.0f + static_cast<float>(rand()) / RAND_MAX;
        } else {
            h_data[i] = 8.0f + static_cast<float>(rand()) / RAND_MAX;
        }
    }

    float h_mu[NUM_CLUSTERS] = {1.0f, 9.0f};
    float h_sigma[NUM_CLUSTERS] = {1.0f, 1.0f};
    float h_pival[NUM_CLUSTERS] = {0.5f, 0.5f};

    float *d_data, *d_mu, *d_sigma, *d_pival;
    float *d_responsibilities, *d_sum_gamma, *d_sum_x, *d_sum_x2;

    CUDA_CHECK(hipMalloc(&d_data, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_mu, NUM_CLUSTERS * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_sigma, NUM_CLUSTERS * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_pival, NUM_CLUSTERS * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_responsibilities, N * NUM_CLUSTERS * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_sum_gamma, NUM_CLUSTERS * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_sum_x, NUM_CLUSTERS * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_sum_x2, NUM_CLUSTERS * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_data, h_data, N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_mu, h_mu, NUM_CLUSTERS * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_sigma, h_sigma, NUM_CLUSTERS * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_pival, h_pival, NUM_CLUSTERS * sizeof(float), hipMemcpyHostToDevice));

    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    float h_sum_gamma[NUM_CLUSTERS];
    float h_sum_x[NUM_CLUSTERS];
    float h_sum_x2[NUM_CLUSTERS];

    int maxIter = 100;
    for (int iter = 0; iter < maxIter; iter++) {

        eStepKernel<<<blocks, THREADS_PER_BLOCK>>>(d_data, N, d_mu, d_sigma, d_pival, d_responsibilities);
        CUDA_CHECK(hipDeviceSynchronize());

        CUDA_CHECK(hipMemset(d_sum_gamma, 0, NUM_CLUSTERS * sizeof(float)));
        CUDA_CHECK(hipMemset(d_sum_x, 0, NUM_CLUSTERS * sizeof(float)));
        CUDA_CHECK(hipMemset(d_sum_x2, 0, NUM_CLUSTERS * sizeof(float)));

        mStepKernel<<<blocks, THREADS_PER_BLOCK>>>(d_data, N, d_responsibilities, d_sum_gamma, d_sum_x, d_sum_x2);
        CUDA_CHECK(hipDeviceSynchronize());

        CUDA_CHECK(hipMemcpy(h_sum_gamma, d_sum_gamma, NUM_CLUSTERS * sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_sum_x, d_sum_x, NUM_CLUSTERS * sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_sum_x2, d_sum_x2, NUM_CLUSTERS * sizeof(float), hipMemcpyDeviceToHost));

        for (int k = 0; k < NUM_CLUSTERS; k++) {
            if (h_sum_gamma[k] > 1e-6f) {
                h_mu[k] = h_sum_x[k] / h_sum_gamma[k];
                float variance = h_sum_x2[k] / h_sum_gamma[k] - h_mu[k] * h_mu[k];
                h_sigma[k] = sqrtf(fmax(variance, 1e-6f));  
                h_pival[k] = h_sum_gamma[k] / N;
            }
        }

        CUDA_CHECK(hipMemcpy(d_mu, h_mu, NUM_CLUSTERS * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_sigma, h_sigma, NUM_CLUSTERS * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_pival, h_pival, NUM_CLUSTERS * sizeof(float), hipMemcpyHostToDevice));

        if (iter % 10 == 0 || iter == maxIter - 1) {
            std::cout << "Iteration " << iter << ":\n";
            for (int k = 0; k < NUM_CLUSTERS; k++) {
                std::cout << "  Cluster " << k << ": "
                          << "mu = " << h_mu[k] << ", "
                          << "sigma = " << h_sigma[k] << ", "
                          << "pi = " << h_pival[k] << std::endl;
            }
            std::cout << std::endl;
        }
    }

    hipFree(d_data);
    hipFree(d_mu);
    hipFree(d_sigma);
    hipFree(d_pival);
    hipFree(d_responsibilities);
    hipFree(d_sum_gamma);
    hipFree(d_sum_x);
    hipFree(d_sum_x2);

    return 0;
}
